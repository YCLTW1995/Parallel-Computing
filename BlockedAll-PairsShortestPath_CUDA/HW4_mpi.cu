#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <mpi.h>



using namespace std ;
// blocking factor
#define B 32

const int INF = 1000000000;
const int V = 20010;
void input(char* inFileName);
void output(char* outFileName);

void block_FW(int rank , int size );
int ceil(int a, int b);
void cal(int Round, int block_start_x, int block_start_y,
        int block_width, int block_height);

int n, m;  // Number of vertices, edges
//static int Dist[V][V];
static int dist[V*V] ;
__global__ void fuckcuda(int *garr , int * data )
{
    __shared__ int sdist[32*32];
    int round = data[0] ;
    int n = data[1] ;
    int x = threadIdx.x;
    int y = threadIdx.y ;
    int i = x + round * 32 ;
    int j = y + round * 32;
    if(i<n && j < n ){
        sdist[x * 32 + y] = garr[i*n+j];
    }
    else sdist[x * 32 + y] =1000000000;

    __syncthreads();
#pragma unroll
    for (int k = 0 ; (k < 32 )&&((round*32+k) < n) ; k ++){
        int tmp = sdist[ x* 32 + k] + sdist[k *32 + y];
        if (tmp < sdist[x *32 + y]) sdist[x *32 + y] = tmp;
        __syncthreads();

    }

    if (i < n && j < n) {

        garr[i*n+j] = sdist[x * 32 + y];

    } 

    __syncthreads();

}
__global__ void phase2_vertical (int * arr , int *data ) 
{
    //  int round = r ;
    int round = data[0] ;
    int n = data[1] ;
    int whichcol = blockIdx.y ;
    if(whichcol != round){

        __shared__ int TMP[64][64];
        __shared__ int DOING[64][64];
        int i = threadIdx.y;
        int j = threadIdx.x;
        int col = blockIdx.y * 32 + i;
        int row = round * 32 + j;
        int start = 32 * round ; 
        int end = 0 ;


        if(col >= n || row >= n){
            DOING[i][j] = INF;
        }else{
            DOING[i][j] = arr[col*n  + row];
        }
        int doingi = start + i;
        int doingj = start + j;
        if(doingi >= n || doingj >= n){
            TMP[i][j] = INF;
        }else{
            TMP[i][j] = arr[doingi*n + doingj] ;
        }

        __syncthreads();
        if(col >= n || row >= n) return;

        if (start + 32 < n ) end = 32 ;
        else end = n - start ;


        for(int k = 0; k < end ; k++ ){
            int check = DOING[i][k] + TMP[k][j] ;
            if(check < DOING[i][j]){
                DOING[i][j] = check ;
            }
        }
        arr[col*n +row] = DOING[i][j] ;
    }
    else {
        return ;
    }


}
__global__ void phase2_row(int * arr , int * data  )
{
    //     int round = r ;
    int round = data[0] ;
    int n = data[1] ;
    int whichrow = blockIdx.x ;
    if(whichrow != round){
        __shared__ int TMP[64][64];
        __shared__ int DOING[64][64];
        int i = threadIdx.y;
        int j = threadIdx.x;
        int col = round * 32 + i;
        int row = blockIdx.x * 32 + j;
        int start = 32 * round ; 
        int end = 0 ;


        if(col >= n || row >= n){
            DOING[i][j] = INF;
        }else{
            DOING[i][j] = arr[col*n  +row];
        }
        int doingi = start + i;
        int doingj = start + j;
        if(doingi >= n || doingj >= n){
            TMP[i][j] = INF;
        }else{
            TMP[i][j] = arr[doingi*n + doingj] ;
        }

        __syncthreads();
        if(col >= n || row >= n) return;

        if (start + 32 < n ) end = 32 ;
        else end = n - start ;

        for(int k = 0; k < end ; k++){
            int check = DOING[k][j] + TMP[i][k] ; 
            if(check < DOING[i][j] ) {
                DOING[i][j] = check ;
            }
        }
        arr[col*n +row ]= DOING[i][j] ;
    }
    else return ;
}

int main(int argc, char* argv[]) {

    int size , rank ;

    MPI_Init(&argc , &argv) ;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank) ;
    MPI_Comm_size(MPI_COMM_WORLD, &size ) ;

    input(argv[1]);
    /*
       for (int i = 0 ; i < n ; i ++ ){
       for (int j = 0 ; j < n ; j ++ ){
       dist[i*n+j] = Dist[i][j];
       }
       }*/
    block_FW(rank , size );
    if(rank == 0 ) {
        output(argv[2]);
    }
    MPI_Finalize() ;
    return 0;
}

void input(char* inFileName) {
    FILE* infile = fopen(inFileName, "r");
    fscanf(infile, "%d %d", &n, &m);

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j)
                dist[i*n+j] = 0;
            else
                dist[i*n+j] = INF;
        }
    }

    while (--m >= 0) {
        int a, b, v;
        fscanf(infile, "%d %d %d", &a, &b, &v);
        dist[a*n+b] = v;
    }
}

void output(char* outFileName) {
    FILE *outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            // if (Dist[i][j] >= INF)	fprintf(outfile, "INF ");
            // else					fprintf(outfile, "%d ", Dist[i][j]);
            if (dist[i*n+j] >= INF)
                dist[i*n+j] = INF;


        }
        fwrite(&dist[i*n], sizeof( int), n, outfile);
    }
    fclose(outfile);
}

int ceil(int a, int b) { return (a + b - 1) / b; }
__global__ void Phase_3(int *device, int round, int n)
{
    if(blockIdx.x == round || blockIdx.y == round) return;
    int start = round*32;
    int end = ((start + 32) < n) ? 32 : n - start;
    int i = threadIdx.y;
    int j = threadIdx.x;
    int col = blockIdx.y * 32 + i;
    int row = blockIdx.x * 32 + j;
    __shared__ int Pi[32][32];
    __shared__ int Pj[32][32];
    if(col >= n || start + j >= n){
        Pi[i][j] = 1000000000 ;
    }
    else{
        Pi[i][j] = device[col*n + (start + j)];
    }
    if(row >= n || start + i >= n){
        Pj[i][j]  =  1000000000 ;
    }
    else {
        Pj[i][j] =  device[(start + i)*n + row];
    }
    //printf("fuck\n");
    if(row < n && col < n) {
        int check = device[col*n + row];
        __syncthreads();
        for(int k = 0; k < end ; ++k){
            check = min(check, Pi[i][k] + Pj[k][j]);
        }
        device[col*n + row] = check;
    }
    else return ;
}
void block_FW(int rank , int size ) {
    int round = ceil(n, B);
    //int r = 1 ;

    dim3 blocknum(1,1) ;
    dim3 blocksize(32,32) ;
    ssize_t arrsize = n*n*sizeof(int) ;

    int *data ;
    data = new int [2] ;

    int *garr ;
    int *gdata ;
    hipMalloc(&garr, arrsize) ;
    hipMalloc(&gdata , sizeof(int)*2) ;

    hipMemcpy(garr,dist, arrsize,hipMemcpyHostToDevice);

    //int *phase2ans1 , *phase2ans2 ;
   // phase2ans1 = new int [n*n ] ;
   // phase2ans2 = new int [n*n ] ;

    MPI_Barrier(MPI_COMM_WORLD) ;
    for (int r = 0; r < round; ++r) {

        /* Phase 1*/

        data[0] = r ;
        data[1] = n ;
    

        hipMemcpy(gdata,data, sizeof(int)*2,hipMemcpyHostToDevice);
        if(rank == 0 ){
   //       hipMemcpy(garr,dist, arrsize,hipMemcpyHostToDevice);
            fuckcuda<<<blocknum, blocksize>>>(garr , gdata);  
        }
        else if(rank == 1 ) {
   //       hipMemcpy(garr,dist, arrsize,hipMemcpyHostToDevice);
   //         fuckcuda<<<blocknum, blocksize>>>(garr , gdata);  
            
        }
    /*    
        if(rank == 1 ) {
            MPI_Send(&dist[pos] ,height * n , MPI_INT , 0 , 0 , MPI_COMM_WORLD ) ;    
        }
        else if(rank == 0 ) {
            MPI_Recv(&dist[pos] , height * n , MPI_INT , 1 , 0 , MPI_COMM_WORLD, MPI_STATUS_IGNORE) ;
        }
        */
    
        
        
        MPI_Barrier(MPI_COMM_WORLD);
       
        
        
        
        // MPI_Bcast( &dist[pos], height * n , MPI_INT, 1 , MPI_COMM_WORLD  ) ;
    //    MPI_Bcast(dist , n*n , MPI_INT , 0 , MPI_COMM_WORLD) ;
   //     hipMemcpy(garr , dist , arrsize , hipMemcpyHostToDevice);
        if(rank == 0 ) {
            
   //       hipMemcpy(garr,dist, arrsize,hipMemcpyHostToDevice);
   //         fuckcuda<<<blocknum, blocksize>>>(garr , gdata);  
            phase2_vertical<<<dim3(1,round), dim3(32,32)>>>(garr , gdata ) ;
            phase2_row<<<dim3(round,1), dim3(32,32)>>>(garr, gdata) ;
            hipMemcpy(dist , garr , arrsize , hipMemcpyDeviceToHost) ;
        }
        else if(rank == 1 ) {
    
   //       hipMemcpy(garr,dist, arrsize,hipMemcpyHostToDevice);
   //         fuckcuda<<<blocknum, blocksize>>>(garr , gdata);  
            phase2_row<<<dim3(round,1), dim3(32,32)>>>(garr, gdata) ;
            hipMemcpy(dist , garr , arrsize , hipMemcpyDeviceToHost) ;

        }
        MPI_Barrier(MPI_COMM_WORLD);
        

        /////Phase3 

        int start = r*32 ;
        int height = 32 ;
        if(start + 32 > n ) height = n - start ;
        int pos = start * n ;
        MPI_Bcast( &dist[pos], height * n , MPI_INT, 0 , MPI_COMM_WORLD  ) ;
    /*    
        if(rank == 1 ) {
            MPI_Send(&dist[pos] ,height * n , MPI_INT , 0 , 0 , MPI_COMM_WORLD ) ;    
        }
        else if(rank == 0 ) {
            MPI_Recv(&dist[pos] , height * n , MPI_INT , 1 , 0 , MPI_COMM_WORLD, MPI_STATUS_IGNORE) ;
        }
        */
        MPI_Barrier(MPI_COMM_WORLD);
       // MPI_Bcast( &dist[pos], height * n , MPI_INT, 1 , MPI_COMM_WORLD  ) ;
    //    MPI_Bcast(dist , n*n , MPI_INT , 0 , MPI_COMM_WORLD) ;
   //     hipMemcpy(garr , dist , arrsize , hipMemcpyHostToDevice);
        if(rank == 0 ) {
        //    hipMemcpy(garr , dist , arrsize , hipMemcpyHostToDevice);
            Phase_3<<<dim3(round, round), dim3(32, 32)>>>(garr, r, n);
        }
        MPI_Barrier(MPI_COMM_WORLD) ;

    }
    if(rank == 0 ){
        hipMemcpy(dist, garr, arrsize, hipMemcpyDeviceToHost);
    }
    hipFree(garr);
    hipFree(gdata) ;
    delete[] data ;
    MPI_Barrier(MPI_COMM_WORLD);
}

