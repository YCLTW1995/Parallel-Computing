#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <omp.h>


using namespace std ;
// blocking factor
#define B 32

const int INF = 1000000000;
const int V = 20010;
void input(char* inFileName);
void output(char* outFileName);

void block_FW();
int ceil(int a, int b);
void cal(int Round, int block_start_x, int block_start_y,
        int block_width, int block_height);

int n, m;  // Number of vertices, edges
//static int Dist[V][V];
static int dist[V*V] ;
__global__ void fuckcuda(int *garr , int * data )
{
    __shared__ int sdist[32*32];
    int round = data[0] ;
    int n = data[1] ;
    int x = threadIdx.x;
    int y = threadIdx.y ;
    int i = x + round * 32 ;
    int j = y + round * 32;
    if(i<n && j < n ){
        sdist[x * 32 + y] = garr[i*n+j];
    }
    else sdist[x * 32 + y] =1000000000;

    __syncthreads();
#pragma unroll
    for (int k = 0 ; (k < 32 )&&((round*32+k) < n) ; k ++){
        int tmp = sdist[ x* 32 + k] + sdist[k *32 + y];
        if (tmp < sdist[x *32 + y]) sdist[x *32 + y] = tmp;
        __syncthreads();

    }

    if (i < n && j < n) {

        garr[i*n+j] = sdist[x * 32 + y];

    } 

    __syncthreads();

}
__global__ void phase2_vertical (int * arr , int *data ) 
{
    //  int round = r ;
    int round = data[0] ;
    int n = data[1] ;
    int whichcol = blockIdx.y ;
    if(whichcol != round){

        __shared__ int TMP[64][64];
        __shared__ int DOING[64][64];
        int i = threadIdx.y;
        int j = threadIdx.x;
        int col = blockIdx.y * 32 + i;
        int row = round * 32 + j;
        int start = 32 * round ; 
        int end = 0 ;


        if(col >= n || row >= n){
            DOING[i][j] = INF;
        }else{
            DOING[i][j] = arr[col*n  + row];
        }
        int doingi = start + i;
        int doingj = start + j;
        if(doingi >= n || doingj >= n){
            TMP[i][j] = INF;
        }else{
            TMP[i][j] = arr[doingi*n + doingj] ;
        }

        __syncthreads();
        if(col >= n || row >= n) return;

        if (start + 32 < n ) end = 32 ;
        else end = n - start ;


        for(int k = 0; k < end ; k++ ){
            int check = DOING[i][k] + TMP[k][j] ;
            if(check < DOING[i][j]){
                DOING[i][j] = check ;
            }
        }
        arr[col*n +row] = DOING[i][j] ;
    }
    else {
        return ;
    }


}
__global__ void phase2_row(int * arr , int * data  )
{
    //     int round = r ;
    int round = data[0] ;
    int n = data[1] ;
    int whichrow = blockIdx.x ;
    if(whichrow != round){
        __shared__ int TMP[64][64];
        __shared__ int DOING[64][64];
        int i = threadIdx.y;
        int j = threadIdx.x;
        int col = round * 32 + i;
        int row = blockIdx.x * 32 + j;
        int start = 32 * round ; 
        int end = 0 ;


        if(col >= n || row >= n){
            DOING[i][j] = INF;
        }else{
            DOING[i][j] = arr[col*n  +row];
        }
        int doingi = start + i;
        int doingj = start + j;
        if(doingi >= n || doingj >= n){
            TMP[i][j] = INF;
        }else{
            TMP[i][j] = arr[doingi*n + doingj] ;
        }

        __syncthreads();
        if(col >= n || row >= n) return;

        if (start + 32 < n ) end = 32 ;
        else end = n - start ;

        for(int k = 0; k < end ; k++){
            int check = DOING[k][j] + TMP[i][k] ; 
            if(check < DOING[i][j] ) {
                DOING[i][j] = check ;
            }
        }
        arr[col*n +row ]= DOING[i][j] ;
    }
    else return ;
}

int main(int argc, char* argv[]) {
    input(argv[1]);
    /*
       for (int i = 0 ; i < n ; i ++ ){
       for (int j = 0 ; j < n ; j ++ ){
       dist[i*n+j] = Dist[i][j];
       }
       }*/
    block_FW();

    output(argv[2]);

    return 0;
}

void input(char* inFileName) {
    FILE* infile = fopen(inFileName, "r");
    fscanf(infile, "%d %d", &n, &m);

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j)
                dist[i*n+j] = 0;
            else
                dist[i*n+j] = INF;
        }
    }

    while (--m >= 0) {
        int a, b, v;
        fscanf(infile, "%d %d %d", &a, &b, &v);
        dist[a*n+b] = v;
    }
}

void output(char* outFileName) {
    FILE *outfile = fopen(outFileName, "w");
     for (int i = 0; i < n; ++i) {
       for (int j = 0; j < n; ++j) {
    // if (Dist[i][j] >= INF)	fprintf(outfile, "INF ");
    // else					fprintf(outfile, "%d ", Dist[i][j]);
    if (dist[i*n+j] >= INF)
    dist[i*n+j] = INF;


    }
    fwrite(&dist[i*n], sizeof( int), n, outfile);
    }
     /*
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (dist[i*n+j] >= INF)
                fprintf(outfile, "INF ");
            else
                fprintf(outfile, "%d ", dist[i*n+j]);
        }
        fprintf(outfile, "\n");
    }
    */
    fclose(outfile);
}

int ceil(int a, int b) { return (a + b - 1) / b; }
__global__ void Phase_3(int *device, int round, int n)
{
    if(blockIdx.x == round || blockIdx.y == round) return;
    int start = round*32;
    int end = ((start + 32) < n) ? 32 : n - start;
    int i = threadIdx.y;
    int j = threadIdx.x;
    int col = blockIdx.y * 32 + i;
    int row = blockIdx.x * 32 + j;
    __shared__ int Pi[32][32];
    __shared__ int Pj[32][32];
    if(col >= n || start + j >= n){
        Pi[i][j] = 1000000000 ;
    }
    else{
        Pi[i][j] = device[col*n + (start + j)];
    }
    if(row >= n || start + i >= n){
        Pj[i][j]  =  1000000000 ;
    }
    else {
        Pj[i][j] =  device[(start + i)*n + row];
    }
    //printf("fuck\n");
    if(row < n && col < n) {
        int check = device[col*n + row];
        __syncthreads();
        for(int k = 0; k < end ; ++k){
            check = min(check, Pi[i][k] + Pj[k][j]);
        }
        device[col*n + row] = check;
    }
    else return ;
}
void block_FW() {
    int round = ceil(n, B);


    int gpunum ;
    hipGetDeviceCount(&gpunum) ;


    dim3 blocknum(1,1) ;
    dim3 blocksize(32,32) ;
    ssize_t arrsize = n*n*sizeof(int) ;

    int *data ;
    data = new int [2] ;
    int *garr ;
    int *garr1;
    int *gdata ;
    int *gdata1 ;
    hipSetDevice(0);
    hipMalloc(&garr, arrsize) ;
    hipMalloc(&gdata , sizeof(int)*2) ;
    hipMemcpy(garr,dist, arrsize,hipMemcpyHostToDevice);

    hipSetDevice(1) ;
    hipMalloc(&garr1, arrsize) ;
    hipMalloc(&gdata1 , sizeof(int)*2) ;
    hipMemcpy(garr1,dist, arrsize,hipMemcpyHostToDevice);
    omp_set_num_threads(2) ;
#pragma omp parallel 
    {

        int cpuid = omp_get_thread_num() ;


        for (int r = 0; r < round; ++r) {


            /* Phase 1*/

            data[0] = r ;
            data[1] = n ;
            if(cpuid == 0 ) {
                hipSetDevice(0) ;

                hipMemcpy(gdata,data, sizeof(int)*2,hipMemcpyHostToDevice);
                fuckcuda<<<blocknum, blocksize>>>(garr , gdata);  
                hipMemcpy(garr1 , garr , arrsize , hipMemcpyDeviceToDevice) ;
            }
            hipDeviceSynchronize() ;
#pragma omp barrier

            //Phase2 
            if(cpuid == 0 ){
                hipSetDevice(0) ;
                phase2_vertical<<<dim3(1,round), dim3(32,32)>>>(garr , gdata ) ;
//                phase2_row<<<dim3(round,1), dim3(32,32)>>>(garr, gdata) ;
            }
            else if(cpuid == 1 ) {
                hipSetDevice(1) ;
                hipMemcpy(gdata1,data, sizeof(int)*2,hipMemcpyHostToDevice);
                phase2_row<<<dim3(round,1), dim3(32,32)>>>(garr1, gdata1) ;
      //          phase2_vertical<<<dim3(1,round), dim3(32,32)>>>(garr1 , gdata1 ) ;
                int start = r * 32 ;
                int height = 32 ;
                if(start + 32 > n ) height = n - start ; 
                ssize_t tmpsize = n*height * sizeof(int) ;
                int pos  = 32 * r * n ;
                hipSetDevice(1) ;
                hipMemcpy(&garr[pos] , &garr1[pos] , tmpsize , hipMemcpyDeviceToDevice) ; 
                
                /*      int start = 32 * r  ; 
                int width =  32 ;
                for(int x = 0 ; x < n  ; x ++ ) {
                    if (start + 32  > n ) {
                        width = n - start  ;
                    }
                    ssize_t tmpsize  = width*sizeof(int) ;
                    int pos = x * n  + start  ;
                    hipSetDevice(1) ;
                    hipMemcpy(&garr[pos], &garr1[pos] ,tmpsize  , hipMemcpyDeviceToDevice ) ;
                }
*/
            }
            //Phase3
            hipDeviceSynchronize() ;
#pragma omp barrier
            if(cpuid == 0 ) {
                hipSetDevice(0) ;
                Phase_3<<<dim3(round, round), dim3(32, 32)>>>(garr, r, n);
            }
            hipDeviceSynchronize() ;
#pragma omp barrier
        }
        if(cpuid == 0 ){
            hipSetDevice(0) ;
            hipMemcpy(dist, garr, arrsize, hipMemcpyDeviceToHost);
        }
    }
}


