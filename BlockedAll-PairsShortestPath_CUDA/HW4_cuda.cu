#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <algorithm>

using namespace std ;
// blocking factor
#define B 32

const int INF = 1000000000;
const int V = 20010;
void input(char* inFileName);
void output(char* outFileName);

void block_FW();
int ceil(int a, int b);
void cal(int Round, int block_start_x, int block_start_y,
        int block_width, int block_height);

int n, m;  // Number of vertices, edges
//static int Dist[V][V];
static int dist[V*V] ;
__global__ void fuckcuda(int *garr , int * data )
{
    __shared__ int sdist[32*32];
    int round = data[0] ;
    int n = data[1] ;
    int x = threadIdx.x;
    int y = threadIdx.y ;
    int i = x + round * 32 ;
    int j = y + round * 32;
    if(i<n && j < n ){
        sdist[x * 32 + y] = garr[i*n+j];
    }
    else sdist[x * 32 + y] =1000000000;

    __syncthreads();
#pragma unroll
    for (int k = 0 ; (k < 32 )&&((round*32+k) < n) ; k ++){
        int tmp = sdist[ x* 32 + k] + sdist[k *32 + y];
        if (tmp < sdist[x *32 + y]) sdist[x *32 + y] = tmp;
        __syncthreads();

    }

    if (i < n && j < n) {

        garr[i*n+j] = sdist[x * 32 + y];

    } 

    __syncthreads();

}
__global__ void phase2_vertical (int * arr , int *data ) 
{
    //  int round = r ;
    int round = data[0] ;
    int n = data[1] ;
    int whichcol = blockIdx.y ;
    if(whichcol != round){

        __shared__ int TMP[64][64];
        __shared__ int DOING[64][64];
        int i = threadIdx.y;
        int j = threadIdx.x;
        int col = blockIdx.y * 32 + i;
        int row = round * 32 + j;
        int start = 32 * round ; 
        int end = 0 ;


        if(col >= n || row >= n){
            DOING[i][j] = INF;
        }else{
            DOING[i][j] = arr[col*n  + row];
        }
        int doingi = start + i;
        int doingj = start + j;
        if(doingi >= n || doingj >= n){
            TMP[i][j] = INF;
        }else{
            TMP[i][j] = arr[doingi*n + doingj] ;
        }

        __syncthreads();
        if(col >= n || row >= n) return;

        if (start + 32 < n ) end = 32 ;
        else end = n - start ;


        for(int k = 0; k < end ; k++ ){
            int check = DOING[i][k] + TMP[k][j] ;
            if(check < DOING[i][j]){
                DOING[i][j] = check ;
            }
        }
        arr[col*n +row] = DOING[i][j] ;
    }
    else {
        return ;
    }


}
__global__ void phase2_row(int * arr , int * data  )
{
    //     int round = r ;
    int round = data[0] ;
    int n = data[1] ;
    int whichrow = blockIdx.x ;
    if(whichrow != round){
        __shared__ int TMP[64][64];
        __shared__ int DOING[64][64];
        int i = threadIdx.y;
        int j = threadIdx.x;
        int col = round * 32 + i;
        int row = blockIdx.x * 32 + j;
        int start = 32 * round ; 
        int end = 0 ;


        if(col >= n || row >= n){
            DOING[i][j] = INF;
        }else{
            DOING[i][j] = arr[col*n  +row];
        }
        int doingi = start + i;
        int doingj = start + j;
        if(doingi >= n || doingj >= n){
            TMP[i][j] = INF;
        }else{
            TMP[i][j] = arr[doingi*n + doingj] ;
        }

        __syncthreads();
        if(col >= n || row >= n) return;

        if (start + 32 < n ) end = 32 ;
        else end = n - start ;

        for(int k = 0; k < end ; k++){
            int check = DOING[k][j] + TMP[i][k] ; 
            if(check < DOING[i][j] ) {
                DOING[i][j] = check ;
            }
        }
        arr[col*n +row ]= DOING[i][j] ;
    }
    else return ;
}

int main(int argc, char* argv[]) {
    input(argv[1]);
/*
    for (int i = 0 ; i < n ; i ++ ){
        for (int j = 0 ; j < n ; j ++ ){
            dist[i*n+j] = Dist[i][j];
        }
    }*/
    block_FW();

    output(argv[2]);

    return 0;
}

void input(char* inFileName) {
    FILE* infile = fopen(inFileName, "r");
    fscanf(infile, "%d %d", &n, &m);

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j)
                dist[i*n+j] = 0;
            else
                dist[i*n+j] = INF;
        }
    }

    while (--m >= 0) {
        int a, b, v;
        fscanf(infile, "%d %d %d", &a, &b, &v);
        dist[a*n+b] = v;
    }
}

void output(char* outFileName) {
FILE *outfile = fopen(outFileName, "w");
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			// if (Dist[i][j] >= INF)	fprintf(outfile, "INF ");
			// else					fprintf(outfile, "%d ", Dist[i][j]);
            if (dist[i*n+j] >= INF)
                dist[i*n+j] = INF;

            
		}
		fwrite(&dist[i*n], sizeof( int), n, outfile);
	}
    fclose(outfile);
}

int ceil(int a, int b) { return (a + b - 1) / b; }
__global__ void Phase_3(int *device, int round, int n)
{
    if(blockIdx.x == round || blockIdx.y == round) return;
    int start = round*32;
    int end = ((start + 32) < n) ? 32 : n - start;
    int i = threadIdx.y;
    int j = threadIdx.x;
    int col = blockIdx.y * 32 + i;
    int row = blockIdx.x * 32 + j;
    __shared__ int Pi[32][32];
    __shared__ int Pj[32][32];
    if(col >= n || start + j >= n){
        Pi[i][j] = 1000000000 ;
    }
    else{
        Pi[i][j] = device[col*n + (start + j)];
    }
    if(row >= n || start + i >= n){
        Pj[i][j]  =  1000000000 ;
    }
    else {
        Pj[i][j] =  device[(start + i)*n + row];
    }
    //printf("fuck\n");
    if(row < n && col < n) {
        int check = device[col*n + row];
        __syncthreads();
        for(int k = 0; k < end ; ++k){
            check = min(check, Pi[i][k] + Pj[k][j]);
        }
        device[col*n + row] = check;
    }
    else return ;
}
void block_FW() {
    int round = ceil(n, B);
    //int r = 1 ;

    dim3 blocknum(1,1) ;
    dim3 blocksize(32,32) ;
    ssize_t arrsize = n*n*sizeof(int) ;

    int *data ;
    data = new int [2] ;

    int *garr ;
    int *gdata ;
    hipMalloc(&garr, arrsize) ;
    hipMalloc(&gdata , sizeof(int)*2) ;

    int *sortans3 ;
    sortans3= new int [n*n] ;
    int *sortans ;
    sortans = new int [n*n] ;
/*
    for (int i = 0 ; i < n ; i ++ ){
        for (int j = 0 ; j < n ; j ++ ){
            dist[i*n+j] = Dist[i][j];
        }
    }*/



        hipMemcpy(garr,dist, arrsize,hipMemcpyHostToDevice);

    for (int r = 0; r < round; ++r) {


        /* Phase 1*/

        data[0] = r ;
        data[1] = n ;
        hipMemcpy(gdata,data, sizeof(int)*2,hipMemcpyHostToDevice);
//        data[0] = r ;
  //      data[1] = n ;

//        cudaMemcpy(garr,dist, arrsize,cudaMemcpyHostToDevice);
  //      cudaMemcpy(gdata,data, sizeof(int)*2,cudaMemcpyHostToDevice);
        fuckcuda<<<blocknum, blocksize>>>(garr , gdata);  


    //    cudaMemcpy(dist, garr, arrsize, cudaMemcpyDeviceToHost);
        /*
           for (int i = 0 ; i < n ; i ++ ){
           for (int j = 0 ; j < n ; j ++){

           Dist[i][j] = sortans[i*n+j] ;

           }
           }
         */
        //cudaFree(garr);
        //cudaFree(gdata) ;
        //delete[] sortans ;
        //delete[] data ;
        /* Phase 2*/
        /*
           for (int i = 0 ; i < n ; i ++ ){
           for (int j = 0 ; j < n ; j ++ ){
           dist[i*n+j] = Dist[i][j];
           }
           }*/
      //  cudaMemcpy(garr,dist, arrsize,cudaMemcpyHostToDevice);
      //  cudaMemcpy(gdata,data, sizeof(int)*2,cudaMemcpyHostToDevice);
        //        cudaMemcpy(gdata,data, sizeof(int)*2,cudaMemcpyHostToDevice);

        phase2_vertical<<<dim3(1,round), dim3(32,32)>>>(garr , gdata ) ;
     //   cudaMemcpy( dist , garr,arrsize , cudaMemcpyDeviceToHost) ;


       // cudaMemcpy(garr,dist, arrsize,cudaMemcpyHostToDevice);
       // cudaMemcpy(gdata,data, sizeof(int)*2,cudaMemcpyHostToDevice);

        phase2_row<<<dim3(round,1), dim3(32,32)>>>(garr, gdata) ;

        //cudaMemcpy(dist , garr,arrsize , cudaMemcpyDeviceToHost) ;


        /*
           cal(r, r, 0, r, 1);
           cal(r, r, r + 1, round - r - 1, 1);
           cal(r, 0, r, 1, r);
           cal(r, r + 1, r, 1, round - r - 1);
         */
        /*
           for (int i = 0 ; i < n ; i ++ ){
           for (int j = 0 ; j < n ; j ++ ){
           dist[i*n+j] = Dist[i][j];
           }
           }
         */

      //  cudaMemcpy(garr ,dist, arrsize,cudaMemcpyHostToDevice);

        Phase_3<<<dim3(round, round), dim3(32, 32)>>>(garr, r, n);



        //cudaMemcpy(dist, garr, arrsize, cudaMemcpyDeviceToHost);
        /*

           for (int i = 0 ; i < n ; i ++ ){
           for (int j = 0 ; j < n ; j ++){

           Dist[i][j] = sortans3[i*n+j] ;

           }
           }
         */
        //cudaFree(shit) ;
        //delete[] sortans3 ;
        /*
           cal(r, 0, 0, r, r);
           cal(r, 0, r + 1, round - r - 1, r);
           cal(r, r + 1, 0, r, round - r - 1);
           cal(r, r + 1, r + 1, round - r - 1, round - r - 1);
         */

    }

    hipMemcpy(dist, garr, arrsize, hipMemcpyDeviceToHost);
/*
    for (int i = 0 ; i < n ; i ++ ){
        for (int j = 0 ; j < n ; j ++){

            Dist[i][j] = dist[i*n+j] ;

        }
    }
    */
    hipFree(garr);
    hipFree(gdata) ;
    delete[] sortans ;
    delete[] data ;
 //   cudaFree(shit) ;
    delete[] sortans3 ;
  //  cudaFree(phase2arr1) ;
  //  cudaFree(phase2arr2) ;
}

